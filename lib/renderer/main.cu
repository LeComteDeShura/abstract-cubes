#include "hip/hip_runtime.h"
// #define STB_IMAGE_IMPLEMENTATION
// #include "stb_image.h"
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"
#include <iostream>
#include <cstdio>
#include <chrono>
#include <fstream>
#include <vector>
#include <sstream>
#include <random>
#include<nlohmann/json.hpp>
#include <unistd.h>  //Для getwd
#include <limits.h>  //Для PATH_MAX
#include <stdio.h>   //Для printf

using json = nlohmann::json;
using namespace std;

#define gpuErr(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

// __device__ int cols;
// __device__ int rows;

int cols1;
int rows1;

// int cols;
// int rows;

class vec2;
class vec3;
class vec4;

class vec2 {
public:
    __host__ __device__ vec2(){}
    __host__ __device__ vec2(float _t)
    : x(_t), y(_t) {}
    __host__ __device__ vec2(float _x, float _y)
    : x(_x), y(_y) {}
    float x = 0;
    float y = 0;
};

class vec4 {
public:
    __host__ __device__ vec4(){}
    // __host__ __device__ vec4()
    // : x(0), y(0), z(0), w(0) {}
    __host__ __device__ vec4(float _t)
    : x(_t), y(_t), z(_t), w(_t) {}
    __host__ __device__ vec4(float _x, float _y, float _z, float _w)
    : x(_x), y(_y), z(_z), w(_w) {}
    float x=0, y=0, z=0, w=0;
};

class vec3 {
public:
    __host__ __device__ vec3(){}
    __host__ __device__ vec3(float _t)
    : x(_t), y(_t), z(_t) {}

    __host__ __device__ vec3(vec4 vec){
        x = vec.x;
        y = vec.y;
        z = vec.z;
    }

    __host__ __device__ vec3(float _x, float _y, float _z)
    : x(_x), y(_y), z(_z) {}
    float x=0, y=0, z=0;
};

std::ostream& operator<< (std::ostream &out, const vec3 &vec)
{
    out << "vec3(" << vec.x << ", " << vec.y << ", " << vec.z << ")";
    return out;
}
std::ostream& operator<< (std::ostream &out, const vec2 &vec)
{
    out << "vec2(" << vec.x << ", " << vec.y << ")";
    return out;
}
std::ostream& operator<< (std::ostream &out, const vec4 &vec)
{
    out << "vec4(" << vec.x << ", " << vec.y << ", " << vec.z << ", " << vec.w << ")";
    return out;
}


__host__ __device__ vec2 fract(vec2 vec) {
    vec.x = modff(vec.x, &vec.x);
    vec.y = modff(vec.y, &vec.y);
    return vec;
}

__host__ __device__ vec3 fract(vec3 vec){
    vec.x = modff(vec.x, &vec.x);
    vec.y = modff(vec.y, &vec.y);
    vec.z = modff(vec.z, &vec.z);
    return vec;
}

__host__ __device__ vec4 fract(vec4 vec) {
    vec.x = modff(vec.x, &vec.x);
    vec.y = modff(vec.y, &vec.y);
    vec.z = modff(vec.z, &vec.z);
    vec.w = modff(vec.w, &vec.w);
    return vec;
}

__host__ __device__ float dot(vec4 _vec1, vec4 _vec2) {
    return _vec1.x*_vec2.x + _vec1.y*_vec2.y + _vec1.z*_vec2.z + _vec1.w*_vec2.w;
}

__host__ __device__ float dot(vec3 _vec1, vec3 _vec2) {
    return _vec1.x*_vec2.x + _vec1.y*_vec2.y + _vec1.z*_vec2.z;
}

__host__ __device__ float dot(vec2 _vec1, vec2 _vec2) {
    return _vec1.x*_vec2.x + _vec1.y*_vec2.y;
}


//////////

__host__ __device__ vec2 operator+ (vec2 _vec1, vec2 _vec2) {
    _vec1.x += _vec2.x;
    _vec1.y += _vec2.y;
    return _vec1;
}

__host__ __device__ vec2 operator- (vec2 _vec1, vec2 _vec2) {
    _vec1.x -= _vec2.x;
    _vec1.y -= _vec2.y;
    return _vec1;
}

__host__ __device__ vec2 operator* (vec2 _vec1, vec2 _vec2) {
    _vec1.x *= _vec2.x;
    _vec1.y *= _vec2.y;
    return _vec1;
}

__host__ __device__ vec2 operator/ (vec2 _vec1, vec2 _vec2) {
    _vec1.x /= _vec2.x;
    _vec1.y /= _vec2.y;
    return _vec1;
}

// --------- //

__host__ __device__ vec3 operator+ (vec3 _vec1, vec3 _vec2) {
    _vec1.x += _vec2.x;
    _vec1.y += _vec2.y;
    _vec1.z += _vec2.z;
    return _vec1;
}

__host__ __device__ vec3 operator- (vec3 _vec1, vec3 _vec2) {
    _vec1.x -= _vec2.x;
    _vec1.y -= _vec2.y;
    _vec1.z -= _vec2.z;
    return _vec1;
}

__host__ __device__ vec3 operator* (vec3 _vec1, vec3 _vec2) {
    _vec1.x *= _vec2.x;
    _vec1.y *= _vec2.y;
    _vec1.z *= _vec2.z;
    return _vec1;
}

__host__ __device__ vec3 operator/ (vec3 _vec1, vec3 _vec2) {
    _vec1.x /= _vec2.x;
    _vec1.y /= _vec2.y;
    _vec1.z /= _vec2.z;
    return _vec1;
}

// --------- //

__host__ __device__ vec4 operator+ (vec4 _vec1, vec4 _vec2) {
    _vec1.x += _vec2.x;
    _vec1.y += _vec2.y;
    _vec1.z += _vec2.z;
    _vec1.w += _vec2.w;
    return _vec1;
}

__host__ __device__ vec4 operator- (vec4 _vec1, vec4 _vec2) {
    _vec1.x -= _vec2.x;
    _vec1.y -= _vec2.y;
    _vec1.z -= _vec2.z;
    _vec1.w -= _vec2.w;
    return _vec1;
}

__host__ __device__ vec4 operator* (vec4 _vec1, vec4 _vec2) {
    _vec1.x *= _vec2.x;
    _vec1.y *= _vec2.y;
    _vec1.z *= _vec2.z;
    _vec1.w *= _vec2.w;
    return _vec1;
}

__host__ __device__ vec4 operator/ (vec4 _vec1, vec4 _vec2) {
    _vec1.x /= _vec2.x;
    _vec1.y /= _vec2.y;
    _vec1.z /= _vec2.z;
    _vec1.w /= _vec2.w;
    return _vec1;
}

// --------- //

__host__ __device__ vec3 operator+ (vec3 _vec1, float n) {
    _vec1.x += n;
    _vec1.y += n;
    _vec1.z += n;
    return _vec1;
}

__host__ __device__ vec3 operator+ (float n, vec3 _vec1) {
    _vec1.x += n;
    _vec1.y += n;
    _vec1.z += n;
    return _vec1;
}

__host__ __device__ vec3 operator- (vec3 _vec1, float n) {
    _vec1.x -= n;
    _vec1.y -= n;
    _vec1.z -= n;
    return _vec1;
}

__host__ __device__ vec3 operator- (float n, vec3 _vec1) {
    _vec1.x = n - _vec1.x;
    _vec1.y = n - _vec1.y;
    _vec1.z = n - _vec1.z;
    return _vec1;
}

__host__ __device__ vec3 operator* (vec3 _vec1, float n) {
    _vec1.x *= n;
    _vec1.y *= n;
    _vec1.z *= n;
    return _vec1;
}

__host__ __device__ vec3 operator* (float n, vec3 _vec1) {
    _vec1.x *= n;
    _vec1.y *= n;
    _vec1.z *= n;
    return _vec1;
}

__host__ __device__ vec3 operator/ (float n, vec3 _vec1) {
    _vec1.x = n / _vec1.x;
    _vec1.y = n / _vec1.y;
    _vec1.z = n / _vec1.z;
    return _vec1;
}

__host__ __device__ vec3 operator/ (vec3 _vec1, float n) {
    _vec1.x /= n;
    _vec1.y /= n;
    _vec1.z /= n;
    return _vec1;
}

// __device__ vec4 R_STATE;

std::vector<std::string> split(std::string s, char delimiter){
   std::vector<std::string> tokens;
   std::string token;
   std::istringstream tokenStream(s);
   while (std::getline(tokenStream, token, delimiter))
   {
      tokens.push_back(token);
   }
   return tokens;
}
/////////

__host__ __device__ vec3 abs(vec3 _vec1) {
    _vec1.x =  fabsf(_vec1.x);
    _vec1.y =  fabsf(_vec1.y);
    _vec1.z =  fabsf(_vec1.z);
    return _vec1;
}

__host__ __device__ vec3 sign(vec3 _vec1) {

    _vec1.x = (0 < _vec1.x) - (_vec1.x < 0);
    _vec1.y = (0 < _vec1.y) - (_vec1.y < 0);
    _vec1.z = (0 < _vec1.z) - (_vec1.z < 0);
    return _vec1;
}

/////////
class Mesh {
public:
    Mesh(){

    }

    void readFileWafefront2(string filename){
        ifstream file(filename);
        string str;
        while(getline(file, str)) {
            auto data = split(str, ' ');
            if (str == ""){
                break;
            }
            if (data[0] == "v"){
                vertices.push_back(vec3(std::stof(data[1]), std::stof(data[2]), std::stof(data[3])));
                // vertices.push_back(vec3(1, 2, 3));
                // vertices1.push_back(1);
            }

            if (data[0] == "vn"){
                normals.push_back(vec3(std::stof(data[1]), std::stof(data[2]), std::stof(data[3])));
            }

            if (data[0] == "f") {
                vector<int> idxs;
                vector<int> idxsn;
                for(int i = 1; i < data.size(); i++){
                    if (data[i] == " " || data[i] == "")
                        continue;
                    auto vec = split(data[i], '/');
                    // auto vec = split(data[i], '/');
                    idxs.push_back(std::stoi(vec[0]));
                    idxsn.push_back(std::stoi(vec[2]));
                }
                for (int i = 1; i < idxs.size()-1; i+=1) {
                    vec3 lol = vec3(idxs[0], idxs[i], idxs[i+1]);
                    indexes.push_back(lol);
                    indexesNormal.push_back(idxsn[i]);
                    // if(i == 1)
                }
            }
        }

        file.close();
    }

    void process(){
        vector<vec3> copyVertices;
        for (auto i : indexes) {

            copyVertices.push_back(vertices[int(i.x-1)]);
            copyVertices.push_back(vertices[int(i.y-1)]);
            copyVertices.push_back(vertices[int(i.z-1)]);
        }
        vertices = copyVertices;

        vector<vec3> copyNormals;
        for (auto i : indexesNormal) {

            copyNormals.push_back(normals[int(i-1)]);
            // copyNormals.push_back(vertices[int(i.y-1)]);
            // copyNormals.push_back(vertices[int(i.z-1)]);
            // copyNormals.push_back(vertices[int(i.)])
        }
        normals = copyNormals;

    }

    std::vector<int> indexesNormal;
    vector<vec3> normals;
    // std::vector<int> vertices1;
    std::vector<vec3> vertices;
    std::vector<vec3> indexes;
};

class DeviceModel{
public:
    int sizeVertices;
    vec3* normalsDevice;
    vec3* verticesDevice;
    vec3 position;
    vec4 color;
};

class Model{
public:
    Model(string filenameMesh, vec3 position, vec4 color){
        this->filenameMesh = filenameMesh;
        this->position = position;
        this->color = color;
    }
    Model(){}
    void loadModel(){
        mesh.readFileWafefront2(filenameMesh);
        mesh.process();
        normalsDevice = new vec3[mesh.vertices.size()/3];
        verticesDevice = new vec3[mesh.vertices.size()];
        this->sizeVertices = mesh.vertices.size();
        std::copy(&mesh.vertices[0], &mesh.vertices[sizeVertices], verticesDevice);
        std::copy(&mesh.normals[0], &mesh.normals[sizeVertices/3], normalsDevice);
        // std::copy(std::begin(&(mesh.normals[0])), std::end(&(mesh.normals[0])), std::begin(normalsDevice));
    }

    DeviceModel getDeviceModel(){
        DeviceModel dm;
        this->sizeModel = sizeof(int) + sizeVertices*sizeof(vec3) + sizeVertices/3*sizeof(vec3) + sizeof(vec3) + sizeof(vec4);
        dm.sizeVertices = this->sizeVertices;
        dm.normalsDevice = normalsDevice;
        dm.verticesDevice = verticesDevice;

        dm.position = position;
        dm.color = color;

        return dm;
    }

    int getSizeModel(){

        return sizeModel;
        // return sizeof(int) + this->sizeVertices*sizeof(vec3) + this->sizeVertices/3*sizeof(vec3);
    }

    vec3* normalsDevice;
    vec3* verticesDevice;
    int sizeVertices;
    int sizeModel;
    string filenameMesh;
    Mesh mesh;
    vec3 position;
    vec4 color;
};


json readJson(string filename){
    std::ifstream file(filename);
    json j;
    string jsonStr = "";

    while (file){
        string kek;
        file >> kek;
        jsonStr += kek;
    }
    file.close();
    j = json::parse(jsonStr);
    return j;
}



__device__ DeviceModel* modelsGlobal;
__device__ int numberModelsGlobal;
// __device__ vec4* color123;
// __device__ vec3* normal123;
// __device__ vec3* it1;
// __device__ vec3* its;
// __device__ int sizeVertices;
// __device__ vec3* normalsDevice;

////////

__host__ __device__ vec3 clamp(vec3 _vec1, float _min, float _max) {

    _vec1.x = fminf(fmaxf(_vec1.x, _min), _max);
    _vec1.y = fminf(fmaxf(_vec1.y, _min), _max);
    _vec1.z = fminf(fmaxf(_vec1.z, _min), _max);
    return _vec1;
}

__host__ __device__ vec3 cross(vec3 _vec1, vec3 _vec2) {

    float x = _vec1.y*_vec2.z - _vec1.z*_vec2.y;
    float y = _vec1.z*_vec2.x - _vec1.x*_vec2.z;
    float z = _vec1.x*_vec2.y - _vec1.y*_vec2.x;
    _vec1.x = x;
    _vec1.y = y;
    _vec1.z = z;
    return _vec1;
}

__host__ float norm3(float x, float y, float z){
    return sqrt(x*x+y*y+z*z);
}

__host__ __device__ vec3 normalize(vec3 _vec1) {

    #ifdef __CUDA_ARCH__
    // device implementation
        float l = norm3df(_vec1.x, _vec1.y, _vec1.z);

        _vec1.x = _vec1.x / l;
        _vec1.y = _vec1.y / l;
        _vec1.z = _vec1.z / l;
        return _vec1;
    #else
    // host implementation

        float l = norm3(_vec1.x, _vec1.y, _vec1.z);
        _vec1.x = _vec1.x / l;
        _vec1.y = _vec1.y / l;
        _vec1.z = _vec1.z / l;
        return _vec1;
    #endif
}

__host__ __device__ vec3 reflect(vec3 _vec1, vec3 n) {
    return _vec1 - 2.0f * dot(n, _vec1) * n;
}

__host__ __device__ vec3 refract(vec3 I, vec3 N, float eta) {
    float k = 1.0 - eta * eta * (1.0 - dot(N, I) * dot(N, I));
    vec3 R;
    if (k < 0.0)
        return R;
    else
        R = eta * I - (eta * dot(N, I) + sqrt(k)) * N;
    return R;
}

__host__ __device__ vec3 mix(vec3 _vec1, vec3 _vec2, float a) {
    return _vec1 * ( 1 - a ) + _vec2 * a;
}

///////////
__device__ void printVec(vec3 v){
    printf("%f, %f, %f\n", v.x, v.y, v.z);
}

__device__ void printVec(vec4 v){
    printf("%f, %f, %f %f\n", v.x, v.y, v.z, v.w);
}

__host__ __device__ vec2 sphIntersect(vec3 ro, vec3 rd, float ra) {
	float b = dot(ro, rd);
	float c = dot(ro, ro) - ra * ra;
	float h = b * b - c;
	if(h < 0.0) return vec2(-1.0);
	h = sqrt(h);
	return vec2(-b - h, -b + h);
}

__host__ __device__ float plaIntersect(vec3 ro, vec3 rd, vec4 p) {
	return -(dot(ro, vec3(p)) + p.w) / dot(rd, vec3(p));
}

__host__ __device__ vec3 getSky(vec3 rd) {
    vec3 light = normalize(vec3(-0.5, 0.75, -1.0));
	vec3 col = vec3(0.3, 0.6, 1.0);
	vec3 sun = vec3(0.95, 0.9, 1.0);
	sun = sun * fmaxf(0.0f, powf(dot(rd, light), 256.0f));
	col = col * fmaxf(0.0f, dot(light, vec3(0.0, 0.0, -1.0)));
	return clamp(sun + col * 0.01f, 0.0f, 1.0f);
}

__host__ __device__ unsigned int TausStep(unsigned int z, int S1, int S2, int S3, unsigned int M){
	unsigned int b = (((z << S1) ^ z) >> S2);
	return (((z & M) << S3) ^ b);
}

__host__ __device__ unsigned int LCGStep(unsigned int z, unsigned int A, unsigned int C){
	return (A * z + C);
}

__host__ __device__ vec2 hash22(vec2 p, vec2 seed1){
	p = p + seed1.x;
	vec3 p3 = fract(vec3(p.x, p.y, p.x) * vec3(.1031, .1030, .0973));
    // printVec(p3);
	p3 = p3 + dot(p3, vec3(p3.y, p3.z, p3.x) + 33.33f);
	return fract(( vec2(p3.x, p3.x) + vec2(p3.y, p3.z) ) * vec2(p3.z, p3.x));
}

__host__ __device__ float random1(vec4* R_STATE){
	R_STATE->x = TausStep(R_STATE->x, 13, 19, 12, (unsigned int)(4294967294));
	R_STATE->y = TausStep(R_STATE->y, 2, 25, 4, (unsigned int)(4294967288));
	R_STATE->z = TausStep(R_STATE->z, 3, 11, 17, (unsigned int)(4294967280));
	R_STATE->w = LCGStep(R_STATE->w, (unsigned int)(1664525), (unsigned int)(1013904223));
	return 2.3283064365387e-10 * float(((unsigned int)(R_STATE->x) ^ (unsigned int)(R_STATE->y) ^ (unsigned int)(R_STATE->z) ^ (unsigned int)(R_STATE->w)));
}

__host__ __device__ vec3 randomOnSphere(vec4* R_STATE) {
	vec3 rand = vec3(random1(R_STATE), random1(R_STATE), random1(R_STATE));
	float theta = rand.x * 2.0 * 3.14159265;
	float v = rand.y;
	float phi = acos(2.0 * v - 1.0);
	float r = pow(rand.z, 1.0 / 3.0);
	float x = r * sin(phi) * cos(theta);
	float y = r * sin(phi) * sin(theta);
	float z = r * cos(phi);
	return vec3(x, y, z);
}

__host__ __device__ vec3 triIntersect(vec3 ro, vec3 rd, vec3 v0, vec3 v1, vec3 v2) {
    vec3 v1v0 = v1 - v0;
    vec3 v2v0 = v2 - v0;
    vec3 rov0 = ro - v0;
    vec3  n = cross( v1v0, v2v0 );
    vec3  q = cross( rov0, rd );
    float d = 1.0/dot( rd, n );
    float u = d*dot( -1*q, v2v0 );
    float v = d*dot(  q, v1v0 );
    float t = d*dot( -1*n, rov0 );
    if( u<0.0 || u>1.0 || v<0.0 || (u+v)>1.0 ) t = -1.0;

    return vec3( t, u, v );
}

__host__ __device__ vec3 triangleNormal(vec3 v1, vec3 v2, vec3 v3){
    vec3 A = v1 - v2;
    vec3 B = v3 - v1;
    return normalize(cross( A, B ));
}

__global__ void castRayTriangles(vec3 ro, vec3 rd, vec4* color123) {
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    //
    // if (j > numberModelsGlobal-1 || i > modelsGlobal[j].sizeVertices){
    //     return;
    // }
    // if (i %3 != 0 || i == 0){
    //     return;
    // }
    //
    // // int count = 0;
    // vec3* norm = modelsGlobal[j].normalsDevice;
    // vec3* vert = modelsGlobal[j].verticesDevice;
    // vec3 triPos = modelsGlobal[j].position;
    // vec3 it1 = triIntersect(ro - triPos, rd, vert[i], vert[i+1], vert[i+2]);
    // vec3* its =
    *color123 = vec4(i, j, i, j);
    // if(it1.x > 0.0 && it1.x < 999) {
        // *color = modelsGlobal[j].color;
        // *normal = -1*norm[i/3];
    // }
}

__host__ __device__ vec4 castRay(vec3* ro, vec3* rd, vec4* R_STATE) {

    float MAX_DIST = 99;
	vec4 col;
	vec2 it;
	vec3 n;
    vec2 minIt = vec2(MAX_DIST);
    vec4 spheresPos[5];
    vec4 spheresColor[5];

	spheresPos[0] = vec4(0, 10, 0, 1);
	spheresPos[1] = vec4(0, -10, 0, 1);
	spheresPos[2] = vec4(10, 0, 0, 1);
	spheresPos[3] = vec4(-10, 0, 0, 1);
    // spheresPos[0] = vec4(2, 0, -1, 2);

	spheresColor[0] = vec4(0.9, 0.1, 0.1, 0.7);
	spheresColor[1] = vec4(0.1, 0.9, 0.1, 0.7);
	spheresColor[2] = vec4(0.1, 0.1, 0.9, 0.7);
	spheresColor[3] = vec4(0.9, 0.9, 0.1, 0.7);
    // spheresColor[0] = vec4(1., 1, 1, -2.0);

	for(int i = 0; i < 4; i++) {
		it = sphIntersect(*ro - vec3(spheresPos[i]), *rd, spheresPos[i].w);
		if(it.x > 0.0 && it.x < minIt.x) {
			minIt = it;
			vec3 itPos = *ro + *rd * it.x;
			n = normalize(itPos - vec3(spheresPos[i]));
			col = spheresColor[i];
		}
	}

    #ifdef __CUDA_ARCH__
    // int max_sizeVertices = 0;
    // for (size_t i = 0; i < numberModelsGlobal; i++) {
    //     if (modelsGlobal[i].sizeVertices > max_sizeVertices) {
    //         max_sizeVertices = modelsGlobal[i].sizeVertices;
    //     }
    // }
    //
    // dim3 grid(numberModelsGlobal, 32);
    // dim3 block(numberModelsGlobal/grid.x+1, max_sizeVertices/grid.y+1);
    // vec3 color;
    // // castRayTriangles<<<grid, block>>>(*ro, *rd);
    // vec4* color123 = new vec4;
    // castRayTriangles<<<grid, block>>>(*ro, *rd, color123);
    // hipDeviceSynchronize();
    // // printVec(*color123);
    // delete color123;
    // col = *color;
    // n = *normal;
    // its = new vec3;
    // it.x = it1->x;
    // it.y = it1->y;
    for (int j = 0; j < numberModelsGlobal; j++) {
        int count = 0;
        vec3* norm = modelsGlobal[j].normalsDevice;
        vec3* vert = modelsGlobal[j].verticesDevice;
        for(int i = 0; i < modelsGlobal[j].sizeVertices; i+=3){
            vec3 triPos = modelsGlobal[j].position;
            // printf("%d \n", i);
            // printf("%f, %f, %f\n", verticesDevice[i].x, verticesDevice[i].y, verticesDevice[i].z);
            vec3 it1 = triIntersect(*ro - triPos, *rd, vert[i], vert[i+1], vert[i+2]);
            if(it1.x > 0.0 && it1.x < minIt.x) {
                minIt.x = it1.x;
                minIt.y = it1.y;
                it.x = it1.x;
                it.y = it1.y;
                col = modelsGlobal[j].color;
                n = -1*norm[count];
                // n = 1*triangleNormal(verticesDevice[i], verticesDevice[i+1], verticesDevice[i+2]);
            }
            count++;
        }
    }
    #else
    #endif

	vec3 planeNormal(0.0, 0.0, -1.0);
	it = vec2(plaIntersect(*ro, *rd, vec4(planeNormal.x, planeNormal.y, planeNormal.z, 1.0)));
	if(it.x > 0.0 && it.x < minIt.x) {
		minIt = it;
		n = planeNormal;
		col = vec4(0.5, 0.25, 0.1, 0.9);
	}

    // minIt = vec2(MAX_DIST);
	if(minIt.x == MAX_DIST) {
        vec3 temp =  getSky(*rd);
        return vec4(temp.x, temp.y, temp.z, -2.0f);
    }


	if(col.w == -2.0f)
        return col;

	vec3 reflected = reflect(*rd, n);

	if(col.w < 0.0f) {
		float fresnel = 1.0f - fabsf(dot(-1 * *rd, n));
		if(random1(R_STATE) - 0.1f < fresnel * fresnel) {
			*rd = reflected;
			return col;
		}
		*ro = *ro + *rd * (minIt.y + 0.001f);
		*rd = refract(*rd, n, 1.0f / (1.0f - col.w));
		return col;
	}

	vec3 itPos = *ro + *rd * it.x;
	vec3 r = randomOnSphere(R_STATE);

	vec3 diffuse = normalize(r * dot(r, n));
	*ro = *ro + *rd * (minIt.x - 0.001f);
	*rd = mix(diffuse, reflected, col.w);
	return col;
}

__host__ __device__ vec3 traceRay(vec3 ro, vec3 rd, vec4* R_STATE) {
	vec3 col(1.0f);
    // printVec(ro);
    float MAX_REF = 8;
    // vec3 *ro1 = new vec3(ro);
    // vec3 *rd1 = new vec3(rd);
    // printVec(*ro1, 0, 0);
	for(int i = 0; i < MAX_REF; i++)
	{
		vec4 refCol = castRay(&ro, &rd, R_STATE);
		col = col * vec3(refCol);
		if(refCol.w == -2.0f){
            // delete ro1;
            // delete rd1;
            return col;
        }
	}
    // delete ro1;
    // delete rd1;
	return vec3(0.0);
}
/////

__device__ char * my_strcpy(char *dest, const char *src){
  int i = 0;
  do {
    dest[i] = src[i];}
  while (src[i++] != 0);
  return dest;
}

__device__ char * my_strcat(char *dest, const char *src){
  int i = 0;
  while (dest[i] != 0) i++;
  my_strcpy(dest+i, src);
  return dest;
}
// Device code // main shader
__device__ vec4 rotate(float a) {
    float s = sinf(a);
    float c = cosf(a);
    return vec4(c, -s, s, c);
}

__host__ __device__ vec3 shader(vec2 pixelPosition, vec2 resolution, vec4 rayOrigin, vec3 partFrame, vec3 prevPartFrame, vec2 seed1, vec2 seed2, int numberFrames, vec2 u_mouse){
    vec2 uv = pixelPosition / (resolution / 2) - 1;
	vec2 uvRes = hash22(uv + 1.0f, seed1) * resolution + resolution;
    // vec2 uvRes;
    uv = vec2(uv.y, uv.x);
    vec4 R_STATE;
	R_STATE.x = (unsigned int)(seed1.x + uvRes.x);
	R_STATE.y = (unsigned int)(seed1.y + uvRes.x);
	R_STATE.z = (unsigned int)(seed2.x + uvRes.y);
	R_STATE.w = (unsigned int)(seed2.y + uvRes.y);
    // printVec(uvRes, 99, 99);

    vec3 rayDirection = normalize(vec3(rayOrigin.w, uv.x, uv.y));
    // vec3 rayDirection = vec3(-0.692740, 0.689188, 0.212440);
    float xd = (rayDirection.x*cosf(u_mouse.x))+(rayDirection.y*-sinf(u_mouse.x));
    float yd = (rayDirection.x*sinf(u_mouse.x))+(rayDirection.y*cosf(u_mouse.x));
    rayDirection.x = xd;
    rayDirection.y = yd;
    // float a = u_mouse.x;
    // float zd = (rayDirection.z*cosf(-a))+(rayDirection.x*-sinf(-a));
    // float xd = (rayDirection.z*sinf(-a))+(rayDirection.x*cosf(-a));
    // rayDirection.x = xd;
    // rayDirection.z = zd;

    // float b = 0.1;
    // float xd = (rayDirection.x*cosf(b))+(rayDirection.y*-sinf(b));
    // float yd = (rayDirection.x*sinf(b))+(rayDirection.y*cosf(b));
    // rayDirection.x = xd;
    // rayDirection.y = yd;
    // rayDirection = normalize(rayDirection);

    // #ifdef __CUDA_ARCH__
    // printVec(rayDirection);
    // #endif
	// rayDirection.x = rayDirection.x * rotate(u_mouse.x);
    // rayDirection.y = rayDirection.y * rotate(u_mouse.x);
    // rayDirection.zx *= rotate(-u_mouse.y);
	vec3 rayOrigin1 = vec3(rayOrigin);

	vec3 col(0.0f);
	int samples = 4;
	for(int i = 0; i < samples; i++) {
		col = col + traceRay(rayOrigin1, rayDirection, &R_STATE);
	}

	col = col / samples;
	float white = 20.0;
	col = col * white * 16.0;
	col = (col * (1.0f + col / white / white)) / (1.0f + col);
    if(col.x > 1){
        col.x = 1;
    }
    if(col.y > 1){
        col.y = 1;
    }
    if(col.z > 1){
        col.z = 1;
    }
    if (numberFrames != 1) {
        col = mix(prevPartFrame, col, 1.0f/numberFrames);
    }

    return col;
}

__device__ void itoa(int number, char* lol, int begin) {
    if(number == 0){
        lol[begin + 0] = '0';
        lol[begin + 1] = '0';
        lol[begin + 2] = '0';
        lol[begin + 3] = ';';
        return;
    }

    int i = 2;
    int part = 0;
    while(number != 0) {
        part = number % 10;
        lol[begin + i] = part + '0';
        number /= 10;
        i--;
    }
    for (i=i; i > 0; i--)
        lol[begin + i] = '0';

    lol[begin + 3] = ';';
}



__global__ void kernel(vec2 resolution, vec4 rayOrigin, vec3* frame, char* stringFrame, vec2 seed1, vec2 seed2, int numberFrame, DeviceModel* models, int numberModels, vec2 u_mouse) {
    int rows = blockIdx.x * blockDim.x + threadIdx.x;
    int cols = blockIdx.y * blockDim.y + threadIdx.y;
    // color123 = new vec4;
    // it1 = new vec3;
    // normal = new vec3;
    modelsGlobal = models;
    numberModelsGlobal = numberModels;


    // if(rows == 0 || cols == 0){
    //     printVec(models[0].position);
    // }

    vec3 partFrame;

    if (cols > int(resolution.y)-1 || rows > int(resolution.x)-1) {
        return;
    }
    vec2 pixelPosition(rows, cols);

    vec3 prevPartFrame = frame[rows* int(resolution.y) + cols];
    partFrame = shader(pixelPosition, resolution, rayOrigin, partFrame, prevPartFrame, seed1, seed2, numberFrame, u_mouse);
    frame[rows* int(resolution.y) + cols] = partFrame;

    //???
    // vec3 color = frame[rows* int(resolution.y) + cols] * 255;
    // int begin = rows * int(resolution.y)*20 + cols*20;
    // int numberColor = rows* int(resolution.y) + cols;
    //
    // stringFrame[begin + 0] = '\E';
    // stringFrame[begin + 1] = '[';
    // stringFrame[begin + 2] = '4';
    // stringFrame[begin + 3] = '8';
    // stringFrame[begin + 4] = ';';
    // stringFrame[begin + 5] = '2';
    // stringFrame[begin + 6] = ';';
    // itoa(int(color.x), stringFrame, begin+7);
    // itoa(int(color.y), stringFrame, begin+11);
    // itoa(int(color.z), stringFrame, begin+15);
    //
    // stringFrame[begin + 18] = 'm';
    // if(cols == resolution.y-1){
    //     stringFrame[begin + 19] = '\n';
    // } else {
    //     stringFrame[begin + 19] = ' ';
    // }

    // stringFrame[20*int(resolution.x)*int(resolution.y)] = '\0';

}


class Timer{
private:
	// Псевдонимы типов используются для удобного доступа к вложенным типам
	using clock_t = std::chrono::high_resolution_clock;
	using second_t = std::chrono::duration<double, std::ratio<1> >;

	std::chrono::time_point<clock_t> m_beg;

public:
	Timer() : m_beg(clock_t::now())
	{
	}

	void reset()
	{
		m_beg = clock_t::now();
	}

	double elapsed() const
	{
		return std::chrono::duration_cast<second_t>(clock_t::now() - m_beg).count();
	}
};

class Window {
public:
    Window(int width, int height, vec4 rayOrigin){
        resolution = vec2(width, height);
        this->rayOrigin = rayOrigin;
        sizeFrame = width * height;
        sizeFrameByte = sizeFrame * sizeof(vec3);
        u_mouse = vec2(0,0);
    }

    void loadModels(string _fileObjects) {
        this->fileObjects = _fileObjects;
        json js;

        bool flag = true;
        while(flag){
            try {
              js = readJson(_fileObjects);
              flag = false;
            } catch (...) {
              flag = true;
            }
        }

        numberModels = 0;
        models.clear();
        for (auto i: js){
            numberModels += 1;
            string filename  = i["mesh"];
            vec3 v1 = vec3(i["position"]["x"], i["position"]["y"], i["position"]["z"]);

            vec4 v2 = vec4(i["color"]["r"], i["color"]["g"], i["color"]["b"], i["color"]["a"]);
            Model model(filename, v1, v2);
            model.loadModel();
            models.push_back(model);
        }
    }

    void loadModelsToCuda() {
        int sizeModels = 0;
        DeviceModel dm;
        hostModels.clear();
        for(auto model : models){
            dm = model.getDeviceModel();
            sizeModels += model.getSizeModel();
            hostModels.push_back(dm);
        }

        gpuErr(hipMalloc((void**)&deviceModels, sizeModels));
        vec3* verticesDevice;
        vec3* normalsDevice;
        // vec3* position;
        // vec4* color;
        for (size_t i = 0; i < numberModels; i++) {
            hipMalloc((void**)&verticesDevice, hostModels[i].sizeVertices*sizeof(vec3));
            hipMalloc((void**)&normalsDevice, hostModels[i].sizeVertices/3*sizeof(vec3));
            // hipMalloc((void**)&position, sizeof(vec3));
            // hipMalloc((void**)&color, sizeof(vec4));

            hipMemcpy(verticesDevice, hostModels[i].verticesDevice, hostModels[i].sizeVertices*sizeof(vec3), hipMemcpyHostToDevice);
            hipMemcpy(normalsDevice, hostModels[i].normalsDevice, hostModels[i].sizeVertices/3*sizeof(vec3), hipMemcpyHostToDevice);
            // hipMemcpy(position, hostModels[i].verticesDevice, hostModels[i].sizeVertices*sizeof(vec3), hipMemcpyHostToDevice);
            // hipMemcpy(color, hostModels[i].normalsDevice,*sizeof(vec3), hipMemcpyHostToDevice);

            hipMemcpy(&(deviceModels[i].verticesDevice), &(verticesDevice), sizeof(deviceModels[i].verticesDevice), hipMemcpyHostToDevice);
            hipMemcpy(&(deviceModels[i].normalsDevice), &(normalsDevice), sizeof(deviceModels[i].normalsDevice), hipMemcpyHostToDevice);
            hipMemcpy(&(deviceModels[i].sizeVertices), &(hostModels[i].sizeVertices), sizeof(deviceModels[i].sizeVertices), hipMemcpyHostToDevice);
            hipMemcpy(&(deviceModels[i].position), &(hostModels[i].position), sizeof(deviceModels[i].position), hipMemcpyHostToDevice);
            hipMemcpy(&(deviceModels[i].color), &(hostModels[i].color), sizeof(deviceModels[i].color), hipMemcpyHostToDevice);
        }
    }

    void init(){
        e2 = std::mt19937(rd());
        frame = new vec3[sizeFrame];
        hipMalloc((void**)&deviceFrame, sizeFrameByte);
    }

    void render() {
        // u_mouse.x += 0.1;
        std::uniform_real_distribution<> dist(0.0f, 1.0f);
        seed1 = vec2((float)dist(e2), (float)dist(e2) * 999.0f);
        seed2 = vec2((float)dist(e2), (float)dist(e2) * 999.0f);
        dim3 grid(32, 32);
        dim3 block(resolution.x/grid.x+1, resolution.y/grid.y+1);
        kernel<<<grid, block>>>(resolution, rayOrigin, deviceFrame, deviceStringFrame, seed1, seed2, n, deviceModels, numberModels, u_mouse);
        n++;
        gpuErr(hipMemcpy(frame, deviceFrame, sizeFrameByte, hipMemcpyDeviceToHost));
    }

    void framePrepare(){
        stringFrame = "";
        for (int i = 0; i < resolution.x; i++) {
            for (int j = 0; j < resolution.y; j++) {
                vec3 color = frame[i*int(resolution.y) + j];
                color = (color * 255.0f);
                // if(color.x > 255){
                //     color.x = 240;
                // }
                // if(color.y > 255){
                //     color.y = 240;
                // }
                // if(color.z > 255){
                //     color.z = 240;
                // }
                // char* img

                stringFrame +=  "\e[48;2;" + to_string(int(color.x)) + ";"
                                     + to_string(int(color.y)) + ";"
                                     + to_string(int(color.z)) + "m ";
            }
            stringFrame += "\n";
        }
    }

    void outputFrame(){
        std::cout << flush;
        std::cout << "\e[1;1H";
        std::cout << stringFrame << endl;
    }

    void outputFrameFile(){
        ofstream file("lib/pipe/image", std::ofstream::out | std::ofstream::trunc);
        file << stringFrame << "\n";
        file.close();
    }

    void getCommand(){
        string command;
        string command2;
        ifstream filer("lib/pipe/command");
        filer >> command;
        filer >> command2;
        filer.close();
        if (command != ""){
            n = 1;


            if(command == "left") {
                u_mouse.x =std::stof(command2);
            }

            if(command == "right") {
                u_mouse.x =std::stof(command2);
            }

            if(command == "up") {
                float x = std::cos(u_mouse.x);
                float y = std::sin(u_mouse.x);

                rayOrigin = rayOrigin + vec4(x, y, 0, 0);
            }

            if(command == "down") {
                float x = std::cos(u_mouse.x);
                float y = std::sin(u_mouse.x);

                rayOrigin = rayOrigin - vec4(x, y, 0, 0);
            }

            if(command == "position") {
                json js = readJson(fileObjects);
                int index = 0;
                for (auto i: js) {
                    hostModels[index].position = vec3(i["position"]["x"], i["position"]["y"], i["position"]["z"]);
                    index ++;
                }
                for (size_t i = 0; i < numberModels; i++) {
                    hipMemcpy(&(deviceModels[i].position), &(hostModels[i].position), sizeof(deviceModels[i].position), hipMemcpyHostToDevice);
                }
            }

            hipFree(deviceModels);
            loadModels(fileObjects);
            loadModelsToCuda();

            if(command == "color") {
                json js = readJson(fileObjects);
                int index = 0;
                for (auto i: js) {
                    hostModels[index].color = vec4(i["color"]["r"], i["color"]["g"], i["color"]["b"], i["color"]["a"]);
                    index ++;
                }
                for (size_t i = 0; i < numberModels; i++) {
                    hipMemcpy(&(deviceModels[i].color), &(hostModels[i].color), sizeof(deviceModels[i].color), hipMemcpyHostToDevice);
                }
            }

            if(command == "exit") {
                hipFree(deviceModels);
                hipFree(deviceStringFrame);
                hipFree(deviceFrame);
                delete [] frame;
                // command = "";
                ofstream filew2("lib/pipe/command", std::ofstream::out | std::ofstream::trunc);
                filew2.close();
                exit(0);
            }

            command = "";
            ofstream filew("lib/pipe/command", std::ofstream::out | std::ofstream::trunc);
            filew.close();
        }
    }

    ~Window(){
        hipFree(deviceModels);
        hipFree(deviceStringFrame);
        hipFree(deviceFrame);
        delete [] frame;
    }

    void renderPicture(string filename){
        int n = 0;
        int x = resolution.x;
        int y = resolution.y;

        char* imge = new char[x*y*3];
        for (size_t i = 0; i < x; i++) {
            for (size_t j = 0; j < y; j+=1) {
                vec3 v = frame[i * y + j]*255.0f;
                if (v.x > 255) {
                    v.x = 255;
                }
                if (v.y > 255) {
                    v.y = 255;
                }
                if (v.z > 255) {
                    v.z = 255;
                }
                imge[n] = v.x;
                imge[n+1] = v.y;
                imge[n+2] = v.z;
                n+=3;
            }
        }

        stbi_write_jpg(filename.c_str(), x, y, 3, imge, 100);
    }

int n = 1;
vec2 u_mouse;
vector<Model> models;
DeviceModel* deviceModels;
int numberModels;

vector<DeviceModel> hostModels;

string fileObjects;
string stringFrame;
char* deviceStringFrame;
vec3* frame;
vec3* deviceFrame;
int sizeFrame;
int sizeFrameByte;
vec2 seed1;
vec2 seed2;
std::random_device rd;
std::mt19937 e2;

vec4 rayOrigin;
vec2 resolution;

};

int main(){

    vec4 rayOrigin(0, 0, 0, 1);
    int x = 50;
    int y = 50*2.6;
    Window window(x, y, rayOrigin);
    window.init();
    window.loadModels("lib/pipe/enemies.json");
    window.loadModelsToCuda();


    while(true){
    // for (size_t i = 0; i < 1; i++) {
        window.render();
        window.framePrepare();
        window.outputFrameFile();
        // window.outputFrame();
        window.getCommand();
    }

    return 0;
}


















//
